#include "hip/hip_runtime.h"
%%writefile temp.cu

/**
*   CS6023: GPU Programming 
*   Assignment 1
*   
*   Please don't change any existing code in this file.
*
*   You can add your code whereever needed. Please add necessary memory APIs
*   for your implementation. Use hipFree() to free up memory as soon as you're
*   done with an allocation. This will ensure that you don't run out of memory 
*   while running large test cases. Use the minimum required memory for your 
*   implementation. DO NOT change the kernel configuration parameters.
*/

#include <chrono>
#include <fstream>
#include <iostream>
#include <hip/hip_runtime.h>

using std::cin;
using std::cout;


__global__
void CalculateHadamardProduct(long int* A, long int* B, int N) {
    // TODO: Write your kernel here
    int id = (blockIdx.x + 1) * threadIdx.x;
    if(id<N*N){
      A[id]*=B[id];
    }
}

__global__
void FindWeightMatrix(long int* A, long int* B, int N) {
    // TODO: Write your kernel here
    int id = (blockIdx.x + 1) * threadIdx.x;
    if(id<N*N){
      if(A[id] < B[id]){
        A[id] = B[id];
      }
    }
}

__global__
void CalculateFinalMatrix(long int* A, long int* B, int N) {    
    // TODO: Write your kernel here
    int id = (blockIdx.x + 1) * threadIdx.x;
    int i_B = id / (2 * N);
    int j_B = id % (2 * N);
    if (id < 4 * N * N) {
        int i_A = (i_B % N);
        int j_A = (j_B % N);
        B[id] = B[id] * A[(i_A * N) + j_A];
    }
}


int main(int argc, char** argv) {


    int N;
    cin >> N;
    long int* A = new long int[N * N];
    long int* B = new long int[N * N];
    long int* C = new long int[N * N];
    long int* D = new long int[2 * N * 2 * N];


    for (long int i = 0; i < N * N; i++) {
        cin >> A[i];
    }

    for (long int i = 0; i < N * N; i++) {
        cin >> B[i];
    }

    for (long int i = 0; i < N * N; i++) {
        cin >> C[i];
    }

    for (long int i = 0; i < 2 * N * 2 * N; i++) {
        cin >> D[i];
    }

    /**
     * 
     * DO NOT CHANGE ANYTHING ABOVE THIS LINE
     * 
    */

    long int* d_A;
    long int* d_B;
    long int* d_C;
    long int* d_D;

    dim3 threadsPerBlock(1024, 1, 1);
    dim3 blocksPerGrid(ceil(N * N / 1024.0), 1, 1);

    hipMalloc(&d_A,sizeof(long int)*N*N);
    hipMemcpy(d_A,A,sizeof(long int)*N*N,hipMemcpyHostToDevice);

    hipMalloc(&d_B,sizeof(long int)*N*N);
    hipMemcpy(d_B,B,sizeof(long int)*N*N,hipMemcpyHostToDevice);

    auto start = std::chrono::high_resolution_clock::now();
    CalculateHadamardProduct<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, N);
    hipDeviceSynchronize();

    hipMemcpy(A,d_A,sizeof(long int)*N*N,hipMemcpyDeviceToHost);

    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> elapsed1 = end - start;
    threadsPerBlock = dim3(32, 32, 1);
    blocksPerGrid = dim3(ceil(N * N / 1024.0), 1, 1);

    hipMalloc(&d_C,sizeof(long int)*N*N);
    hipMemcpy(d_C,C,sizeof(long int)*N*N,hipMemcpyHostToDevice);

    start = std::chrono::high_resolution_clock::now();
    FindWeightMatrix<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_C, N);
    hipDeviceSynchronize();

    hipMemcpy(A,d_A,sizeof(long int)*N*N,hipMemcpyDeviceToHost);

    end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> elapsed2 = end - start;


    threadsPerBlock = dim3(32, 32, 1);
    blocksPerGrid = dim3(ceil(2 * N / 32.0), ceil(2 * N / 32.0), 1);

    hipMalloc(&d_D,sizeof(long int)*N*N*4);
    hipMemcpy(d_D,D,sizeof(long int)*N*N*4,hipMemcpyHostToDevice);

    start = std::chrono::high_resolution_clock::now();
    CalculateFinalMatrix<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_D, N);
    hipDeviceSynchronize();
    end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> elapsed3 = end - start;

    hipMemcpy(D,d_D,sizeof(long int)*N*N*4,hipMemcpyDeviceToHost);

    // Make sure your final output from the device is stored in d_D.

    /**
     * 
     * DO NOT CHANGE ANYTHING BELOW THIS LINE
     * 
    */

    hipMemcpy(D, d_D, 2 * N * 2 * N * sizeof(long int), hipMemcpyDeviceToHost);

    std::ofstream file("cuda.out");
    if (file.is_open()) {
        for (long int i = 0; i < 2 * N; i++) {
            for (long int j = 0; j < 2 * N; j++) {
                file << D[i * 2 * N + j] << " ";
            }
            file << "\n";
        }
        file.close();
    } else {
        std::cout << "Unable to open file";
    }

    std::ofstream file2("cuda_timing.out");
    if(file2.is_open()) {
        file2 << elapsed1.count() << "\n";
        file2 << elapsed2.count() << "\n";
        file2 << elapsed3.count() << "\n";
        file2.close();
    } else {
        std::cout << "Unable to open file";
    }

    return 0;
}